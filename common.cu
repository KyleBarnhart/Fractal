#include "hip/hip_runtime.h"
/*
 *  This Quickselect routine is based on the algorithm described in
 *  "Numerical recipes in C", Second Edition,
 *  Cambridge University Press, 1992, Section 8.5, ISBN 0-521-43108-5
 *  This code by Nicolas Devillard - 1998. Public domain.
 *
 * http://ndevilla.free.fr/median/median/index.html
 */
#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include ""

#include "common.h"

void displayCudeError(hipError_t error) {
	std::cerr << hipGetErrorString(error) << std::endl;
   std::cerr << "Press Enter to quit." << std::endl;
	std::cin.ignore();
	exit((int)error);
}

void safeCudaMalloc(void** devPtr, size_t size)
{
   hipError_t error;
   error = hipMalloc(devPtr, size);
   if(error != hipSuccess)
		displayCudeError(error);
}

void safeCudaMemcpy(void* dist, const void* src, size_t size, hipMemcpyKind kind)
{
   hipError_t error;
   error = hipMemcpy(dist, src, size, kind);
   if(error != hipSuccess)
		displayCudeError(error);
}

void safeCudaMemset(void* devPtr, int vlaue, size_t count)
{
   hipError_t error;
   error = hipMemset(devPtr, vlaue, count);
   if(error != hipSuccess)
		displayCudeError(error);
}

void safeCudaFree(void* devPt)
{
   hipError_t error;
   error = hipFree(devPt);
   if(error != hipSuccess)
		displayCudeError(error);
}

__device__ void getRGB(ElementType value, BYTE* rgb)
{ 
   short colourInt = (short)(value * 1792.0f);
   
   BYTE bracket = colourInt / 256;
   BYTE colour = (BYTE)(colourInt % 256);

   switch (bracket)
   {
      case 0:
         rgb[0] = colour;
         rgb[1] = 0;
         rgb[2] = 0;
         break;
          
      case 1:
         rgb[0] = 255;
         rgb[1] = colour;
         rgb[2] = 0;
         break;
          
      case 2:
         rgb[0] = 255 - colour;
         rgb[1] = 255;
         rgb[2] = 0;
         break;

      case 3:
         rgb[0] = 0;
         rgb[1] = 255;
         rgb[2] = colour;
          break;

      case 4:
         rgb[0] = 0;
         rgb[1] = 255 - colour;
         rgb[2] = 255;
         break;

      case 5:
         rgb[0] = colour;
         rgb[1] = 0;
         rgb[2] = 255;
         break;

      case 6:
         rgb[0] = 255 - colour;
         rgb[1] = 0;
         rgb[2] = 255 - colour;
         break;

      default:
         rgb[0] = 0;
         rgb[1] = 0;
         rgb[2] = 0;
          break;    
   }
}

__global__ void getBmpRGB(BYTE* image, ElementType* values, DimensionType width, DimensionType height, IterationType iterations)
{
   DimensionType dy = blockIdx.y * BLOCK_SIZE_RGB + threadIdx.y;  
   DimensionType dx = blockIdx.x * BLOCK_SIZE_RGB + threadIdx.x;
   
   if(dx >= width || dy >= height)
      return; 

   DimensionType c = dy * width + dx;
   
   BYTE rgbValue[3];

   getRGB(values[c]/(ElementType)iterations, rgbValue);
      
   image[c*3]      = rgbValue[2];
   image[c*3 + 1]  = rgbValue[1];
   image[c*3 + 2]  = rgbValue[0];
}

__global__ void getBmpRGBfromHistorgram(ElementType* map, BYTE* image, ElementType* values, DimensionType width, DimensionType height)
{
   DimensionType dy = blockIdx.y * BLOCK_SIZE_RGB + threadIdx.y;  
   DimensionType dx = blockIdx.x * BLOCK_SIZE_RGB + threadIdx.x;
   
   if(dx >= width || dy >= height)
      return; 

   DimensionType c = dy * width + dx;

   IterationType ival = (IterationType)values[c];

   ElementType colourVal = map[ival] + (values[c] - (ElementType)ival) * (map[ival + 1] - map[ival]);
   
   BYTE rgbValue[3];

   getRGB(colourVal, rgbValue);
      
   image[c*3]      = rgbValue[2];
   image[c*3 + 1]  = rgbValue[1];
   image[c*3 + 2]  = rgbValue[0];
}

void histogramToColourMap(DimensionSqType* histogram, ElementType* map, IterationType iterations, DimensionSqType resolution)
{
   ElementType res = (ElementType)resolution;
      
   map[0] = 0.0;

   // Map colors to pixels based on the histogram
   for(IterationType i = 1; i < iterations + 1; i++)
   {
      map[i] = map[i-1] + (ElementType)histogram[i] / res;
   }
}

void valueToRGB(ElementType* values, BYTE* image, IterationType iterations, DimensionType width, DimensionType height)
{
   hipError_t error;
   
   DimensionSqType resolution = (DimensionSqType)width * (DimensionSqType)height;

   // Bytes
   BYTE* deviceBytes;
   safeCudaMalloc((void**)&deviceBytes, resolution * 3);

   // Array of floats for the GPU
   ElementType* deviceValues;
   safeCudaMalloc((void**)&deviceValues, resolution * (DimensionSqType)sizeof(ElementType));

   safeCudaMemcpy(deviceValues, values, resolution * (DimensionSqType)sizeof(ElementType), hipMemcpyHostToDevice);

   // Run fractal on GPU
   int gridWidth = (width / BLOCK_SIZE_RGB) + (width % BLOCK_SIZE_RGB > 0 ? 1 : 0);
   int gridHeight =  (height / BLOCK_SIZE_RGB) + (height % BLOCK_SIZE_RGB > 0 ? 1 : 0);

   dim3 dimBlock(BLOCK_SIZE_RGB, BLOCK_SIZE_RGB);
   dim3 dimGrid(gridWidth, gridHeight);

   getBmpRGB<<<dimGrid, dimBlock>>>(deviceBytes, deviceValues, width, height, iterations);
   if ((error = hipGetLastError()) != hipSuccess)
		displayCudeError(error);

   safeCudaFree(deviceValues);

   // Get fractal values from GPU
   safeCudaMemcpy(image, deviceBytes, resolution * 3, hipMemcpyDeviceToHost);

   safeCudaFree(deviceBytes);
}

void mapValueToRGB(ElementType* map, ElementType* values, BYTE* image, IterationType iterations, DimensionType width, DimensionType height)
{
   hipError_t error;
   
   DimensionSqType resolution = (DimensionSqType)width * (DimensionSqType)height;

   // Map
   ElementType* deviceMapValues;
   safeCudaMalloc((void**)&deviceMapValues, (iterations + 1) * sizeof(ElementType));

   safeCudaMemcpy(deviceMapValues, map, (iterations + 1) * sizeof(ElementType), hipMemcpyHostToDevice);

   // Bytes
   BYTE* deviceBytes;
   safeCudaMalloc((void**)&deviceBytes, resolution * 3);

   // Array of floats for the GPU
   ElementType* deviceValues;
   safeCudaMalloc((void**)&deviceValues, resolution * (DimensionSqType)sizeof(ElementType));

   safeCudaMemcpy(deviceValues, values, resolution * (DimensionSqType)sizeof(ElementType), hipMemcpyHostToDevice);

   // Run fractal on GPU
   int gridWidth = (width / BLOCK_SIZE_RGB) + (width % BLOCK_SIZE_RGB > 0 ? 1 : 0);
   int gridHeight =  (height / BLOCK_SIZE_RGB) + (height % BLOCK_SIZE_RGB > 0 ? 1 : 0);

   dim3 dimBlock(BLOCK_SIZE_RGB, BLOCK_SIZE_RGB);
   dim3 dimGrid(gridWidth, gridHeight);

   getBmpRGBfromHistorgram<<<dimGrid, dimBlock>>>(deviceMapValues, deviceBytes, deviceValues, width, height);
   if ((error = hipGetLastError()) != hipSuccess)
		displayCudeError(error);

   // Get fractal values from GPU
   safeCudaMemcpy(image, deviceBytes, resolution * 3, hipMemcpyDeviceToHost);

   safeCudaFree(deviceMapValues);
   safeCudaFree(deviceBytes);
   safeCudaFree(deviceValues);
}